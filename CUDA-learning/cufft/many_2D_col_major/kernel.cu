#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define NX 3335 // ��Ч���ݸ���
#define NXWITH0 5000
#define Nfft 128
#define BLOCK_SIZE 32
using std::cout;
using std::endl;


/**
* ���ܣ��ж����� cufftComplex ������Ƿ����
* ���룺idataA ��������A��ͷָ��
* ���룺idataB �������B��ͷָ��
* ���룺size �����Ԫ�ظ���
* ���أ�true | false
*/
bool IsEqual(hipfftComplex *idataA, hipfftComplex *idataB, const int size)
{
	for (int i = 0; i < size; i++)
	{
		if (abs(idataA[i].x - idataB[i].x) > 0.000001 || abs(idataA[i].y - idataB[i].y) > 0.000001)
			return false;
	}

	return true;
}


/**
* ���ܣ�ʵ�� cufftComplex ����ĳ߶����ţ�Ҳ���ǳ���һ����
* ���룺idata ���������ͷָ��
* �����odata ��������ͷָ��
* ���룺size �����Ԫ�ظ���
* ���룺scale ���ų߶�
*/
__global__ void CufftComplexScale(hipfftComplex *idata, hipfftComplex *odata, float scale)
{
	// ��������
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * NXWITH0 + x_id;

	if (x_id < NX && y_id < Nfft)
	{
		odata[index].x = idata[index].x * scale;
		odata[index].y = idata[index].y * scale;
	}
}

int main()
{
	/* �����豸�˵��ڴ�ռ� */
	hipfftComplex *data_dev;
	hipMalloc((void**)&data_dev, Nfft * NXWITH0 * sizeof(hipfftComplex)); // �����豸�ڴ�
	hipMemset(data_dev, 0, Nfft * NXWITH0 * sizeof(hipfftComplex)); // ��ʼΪ0

	/* ���������˵��ڴ�ռ� */
	hipfftComplex *data_Host = (hipfftComplex*)malloc(Nfft * NXWITH0 * sizeof(hipfftComplex)); // ����������ͷָ��
	hipfftComplex *resultIFFT = (hipfftComplex*)malloc(Nfft * NXWITH0 * sizeof(hipfftComplex)); // �����任����任�Ľ��


	/* ��ʼ���� */
	for (int i = 0; i < Nfft; i++)
	{
		for (int j = 0; j < NXWITH0; j++)
		{
			int index = i * NXWITH0 + j;
			data_Host[index].x = float((rand() * rand()) % NX) / NX;
			data_Host[index].y = float((rand() * rand()) % NX) / NX;
		}
	}

	/* �߳̿���̸߳� */
	dim3 dimBlock2D(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid2D_NXWITH0_Nfft((NXWITH0 + BLOCK_SIZE - 1) / dimBlock2D.x, (Nfft + BLOCK_SIZE - 1) / dimBlock2D.y);

	/* ���� cufft ���*/
	hipfftHandle plan_Nfft_Many; // ����cuFFT���
	const int rank = 1; // һά fft
	int n[rank] = { Nfft }; // ���� fft ���źŵĳ���Ϊ Nfft
	int inembed[1] = { 0 }; // �������ݵ�[ҳ��������������]
	int onembed[1] = { 10 }; // ������ݵ�[ҳ��������������]
	int istride = NXWITH0; // ÿ�������ź���������Ԫ�صľ���
	int idist = 1; // ÿ���������źŵ�һ��Ԫ�صľ���
	int ostride = NXWITH0; // ÿ������ź���������Ԫ�صľ���
	int odist = 1; // ÿ��������źŵ�һ��Ԫ�صľ���
	int batch = NX; // ���� fft ���źŸ���
	hipfftPlanMany(&plan_Nfft_Many, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

	/* ���Ĳ��� */

	hipMemcpy(data_dev, data_Host, Nfft * NXWITH0 * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftExecC2C(plan_Nfft_Many, data_dev, data_dev, HIPFFT_FORWARD); // ִ�� cuFFT�����任
	hipfftExecC2C(plan_Nfft_Many, data_dev, data_dev, HIPFFT_BACKWARD); // ִ�� cuFFT����任
	CufftComplexScale<<<dimGrid2D_NXWITH0_Nfft, dimBlock2D>>>(data_dev, data_dev, 1.0f / Nfft); // ����ϵ��
	hipMemcpy(resultIFFT, data_dev, Nfft * NXWITH0 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);


	/* ���پ�� */
	hipfftDestroy(plan_Nfft_Many);

	/* �ͷ��豸�ռ� */
	hipFree(data_dev);


	/* ������ */
	cout << IsEqual(data_Host, resultIFFT, NX * Nfft) << endl;

	return 0;
}