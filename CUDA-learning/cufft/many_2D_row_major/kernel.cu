#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define NX 3335 // ��Ч���ݸ���
#define Nfft 128
#define BLOCK_SIZE 32
using std::cout;
using std::endl;


/**
* ���ܣ��ж����� cufftComplex ������Ƿ����
* ���룺idataA ��������A��ͷָ��
* ���룺idataB �������B��ͷָ��
* ���룺size �����Ԫ�ظ���
* ���أ�true | false
*/
bool IsEqual(hipfftComplex *idataA, hipfftComplex *idataB, const int size)
{
	for (int i = 0; i < size; i++)
	{
		if (abs(idataA[i].x - idataB[i].x) > 0.000001 || abs(idataA[i].y - idataB[i].y) > 0.000001)
			return false;
	}

	return true;
}



/**
* ���ܣ�ʵ�� cufftComplex ����ĳ߶����ţ�Ҳ���ǳ���һ����
* ���룺idata ���������ͷָ��
* �����odata ��������ͷָ��
* ���룺size �����Ԫ�ظ���
* ���룺scale ���ų߶�
*/
__global__ void CufftComplexScale(hipfftComplex *idata, hipfftComplex *odata, float scale)
{
	// ��������
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * NX + x_id;

	if (x_id < NX && y_id < Nfft)
	{
		odata[index].x = idata[index].x * scale;
		odata[index].y = idata[index].y * scale;
	}
}

int main()
{
	/* �����豸�˵��ڴ�ռ� */
	hipfftComplex *data_dev;
	hipMalloc((void**)&data_dev, Nfft * NX * sizeof(hipfftComplex)); // �����豸�ڴ�
	hipMemset(data_dev, 0, Nfft * NX * sizeof(hipfftComplex)); // ��ʼΪ0

	/* ���������˵��ڴ�ռ� */
	hipfftComplex *data_Host = (hipfftComplex*)malloc(Nfft * NX * sizeof(hipfftComplex)); // ����������ͷָ��
	hipfftComplex *resultFFT = (hipfftComplex*)malloc(Nfft * NX * sizeof(hipfftComplex)); // ���任�Ľ��
	hipfftComplex *resultIFFT = (hipfftComplex*)malloc(Nfft * NX * sizeof(hipfftComplex)); // �����任����任�Ľ��


	/* ��ʼ���� */
	for (int i = 0; i < Nfft * NX; i++)
	{
		data_Host[i].x = float((rand() * rand()) % NX) / NX;
		data_Host[i].y = float((rand() * rand()) % NX) / NX;
	}

	/* �߳̿���̸߳� */
	dim3 dimBlock2D(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid2D_NX_Nfft((NX + BLOCK_SIZE - 1) / dimBlock2D.x, (Nfft + BLOCK_SIZE - 1) / dimBlock2D.y);

	/* ���� cufft ���*/
	hipfftHandle plan_NX_Many; // ����cuFFT���
	int number_NX[1] = { NX };
	hipfftPlanMany(&plan_NX_Many, 1, number_NX, NULL, 1, NX, NULL, 1, NX, HIPFFT_C2C, Nfft);

	/* ���Ĳ��� */

	hipMemcpy(data_dev, data_Host, Nfft * NX * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftExecC2C(plan_NX_Many, data_dev, data_dev, HIPFFT_FORWARD); // ִ�� cuFFT�����任
	hipfftExecC2C(plan_NX_Many, data_dev, data_dev, HIPFFT_BACKWARD); // ִ�� cuFFT����任
	CufftComplexScale<<<dimGrid2D_NX_Nfft, dimBlock2D>>>(data_dev, data_dev, 1.0f / NX); // ����ϵ��
	hipMemcpy(resultIFFT, data_dev, Nfft * NX * sizeof(hipfftComplex), hipMemcpyDeviceToHost);


	/* ���پ�� */
	hipfftDestroy(plan_NX_Many);

	/* �ͷ��豸�ռ� */
	hipFree(data_dev);


	/* ������ */
	cout << IsEqual(data_Host, resultIFFT, NX * Nfft) << endl;

	return 0;
}