#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"

#include <hipfft/hipfft.h>

#define NX 3335 // ��Ч���ݸ���
#define N 5335 // ��0֮������ݳ���
#define BATCH 1
#define BLOCK_SIZE 1024
using std::cout;
using std::endl;


/**
* ���ܣ��ж����� cufftComplex ������Ƿ����
* ���룺idataA ��������A��ͷָ��
* ���룺idataB �������B��ͷָ��
* ���룺size �����Ԫ�ظ���
* ���أ�true | false
*/
bool IsEqual(hipfftComplex *idataA, hipfftComplex *idataB, const int size)
{
	for (int i = 0; i < size; i++)
	{
		if (abs(idataA[i].x - idataB[i].x) > 0.000001 || abs(idataA[i].y - idataB[i].y) > 0.000001)
			return false;
	}

	return true;
}



/**
* ���ܣ�ʵ�� cufftComplex ����ĳ߶����ţ�Ҳ���ǳ���һ����
* ���룺idata ���������ͷָ��
* �����odata ��������ͷָ��
* ���룺size �����Ԫ�ظ���
* ���룺scale ���ų߶�
*/
static __global__ void cufftComplexScale(hipfftComplex *idata, hipfftComplex *odata, const int size, float scale)
{
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadID < size)
	{
		odata[threadID].x = idata[threadID].x * scale;
		odata[threadID].y = idata[threadID].y * scale;
	}
}

int main()
{
	hipfftComplex *data_dev; // �豸������ͷָ��
	hipfftComplex *data_Host = (hipfftComplex*)malloc(NX*BATCH*sizeof(hipfftComplex)); // ����������ͷָ��
	hipfftComplex *resultFFT = (hipfftComplex*)malloc(N*BATCH*sizeof(hipfftComplex)); // ���任�Ľ��
	hipfftComplex *resultIFFT = (hipfftComplex*)malloc(NX*BATCH*sizeof(hipfftComplex)); // �����任����任�Ľ��

	// ��ʼ����
	for (int i = 0; i < NX; i++)
	{
		data_Host[i].x = float((rand() * rand()) % NX) / NX;
		data_Host[i].y = float((rand() * rand()) % NX) / NX;
	}

	
	dim3 dimBlock(BLOCK_SIZE); // �߳̿�
	dim3 dimGrid((NX + BLOCK_SIZE - 1) / dimBlock.x); // �̸߳�

	hipfftHandle plan; // ����cuFFT���
	hipfftPlan1d(&plan, N, HIPFFT_C2C, BATCH);

	// ��ʱ
	clock_t start, stop;
	double duration;
	start = clock();

	hipMalloc((void**)&data_dev, sizeof(hipfftComplex)*N*BATCH); // �����豸�ڴ�
	hipMemset(data_dev, 0, sizeof(hipfftComplex)*N*BATCH); // ��ʼΪ0
	hipMemcpy(data_dev, data_Host, NX *sizeof(hipfftComplex), hipMemcpyHostToDevice); // �������ڴ濽�����豸�ڴ�

	hipfftExecC2C(plan, data_dev, data_dev, HIPFFT_FORWARD); // ִ�� cuFFT�����任
	hipMemcpy(resultFFT, data_dev, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost); // ���豸�ڴ濽���������ڴ�

	hipfftExecC2C(plan, data_dev, data_dev, HIPFFT_BACKWARD); // ִ�� cuFFT����任
	cufftComplexScale<<<dimGrid, dimBlock>>>(data_dev, data_dev, N, 1.0f / N); // ����ϵ��
	hipMemcpy(resultIFFT, data_dev, NX * sizeof(hipfftComplex), hipMemcpyDeviceToHost); // ���豸�ڴ濽���������ڴ�

	stop = clock();
	duration = (double)(stop-start)*1000/CLOCKS_PER_SEC;
	cout << "ʱ��Ϊ " << duration << " ms" << endl;

	hipfftDestroy(plan); // ���پ��
	hipFree(data_dev); // �ͷſռ�

	cout << IsEqual(data_Host, resultIFFT, NX) << endl;

	return 0;
}