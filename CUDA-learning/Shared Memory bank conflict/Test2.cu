#include "hip/hip_runtime.h"
#include "Test2.h"
#include "IsEqualMatrix.h"

#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#define BLOCKSIZE 32

/**
* ���ܣ�ÿ�� warp ���ʲ�ͬ�� bank����ͬ�� warp ����ͬһ�� bank
* ���룺matrix �������
* �����matrixTest �������
* ���룺row ����
* ���룺col ����
*/
__global__ void SharedMemoryBankConflictTest2(float *matrix, float *matrixTest, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	__shared__ float sData[BLOCKSIZE][BLOCKSIZE];

	if (x_id < col && y_id < row)
	{
		sData[threadIdx.x][threadIdx.y] = matrix[index];
		__syncthreads();

		for (int i = 0; i < 1; i++)
		{
			matrixTest[index] = sData[threadIdx.x][threadIdx.y];
		}
	}
}

/**
* ���ܣ�ÿ�� warp ���ʲ�ͬ�� bank
* ˵����ʹ�õĶ���Ŀռ�
* ���룺matrix �������
* �����matrixTest �������
* ���룺row ����
* ���룺col ����
*/
__global__ void SharedMemoryBankConflictTest2_Pad(float *matrix, float *matrixTest, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	__shared__ float sData[BLOCKSIZE][BLOCKSIZE+1];


	if (x_id < col && y_id < row)
	{
		sData[threadIdx.x][threadIdx.y] = matrix[index];
		__syncthreads();

		for (int i = 0; i < 1; i++)
		{
			matrixTest[index] = sData[threadIdx.x][threadIdx.y];
		}
	}
}

/**
* ���ܣ����棬ͬһ�� warp ����ͬһ�� bank �Ĳ�ֵͬ����ͬ warp ���ʲ�ͬ bank
* ���룺matrix �������
* �����matrixTest �������
* ���룺row ����
* ���룺col ����
*/
__global__ void SharedMemoryBankConflictTest2_Across(float *matrix, float *matrixTest, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	__shared__ float sData[BLOCKSIZE][BLOCKSIZE];


	if (x_id < col && y_id < row)
	{
		sData[threadIdx.y][threadIdx.x] = matrix[index];
		__syncthreads();


		for (int i = 0; i < 1; i++)
		{
			matrixTest[index] = sData[threadIdx.y][threadIdx.x];
		}
	}
}

/**
* ���ܣ��ڶ�������ʾ�������棬ͬһ�� warp ����ͬһ�� bank �Ĳ�ֵͬ
* ���룺matrix_Dev �������
* ���룺row ����
* ���룺col ����
*/
void Test2(float *matrix_Dev, const int row, const int col)
{
	float *matrixTestA_Dev, *matrixTestB_Dev, *matrixTestC_Dev;
	hipMalloc((void**)&matrixTestA_Dev, row * col * sizeof(float));
	hipMemset(matrixTestA_Dev, 0, row * col * sizeof(float));
	hipMalloc((void**)&matrixTestB_Dev, row * col * sizeof(float));
	hipMemset(matrixTestB_Dev, 0, row * col * sizeof(float));
	hipMalloc((void**)&matrixTestC_Dev, row * col * sizeof(float));
	hipMemset(matrixTestC_Dev, 0, row * col * sizeof(float));

	float *matrixTestA_Host = (float *)malloc(row * col * sizeof(float));
	float *matrixTestB_Host = (float *)malloc(row * col * sizeof(float));
	float *matrixTestC_Host = (float *)malloc(row * col * sizeof(float));

	dim3 dimBlock2D(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid2D((col+ BLOCKSIZE - 1) / dimBlock2D.x, (row + BLOCKSIZE - 1) / dimBlock2D.y);

	// ��¼ʱ��
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	hipEventRecord(start_GPU, 0);

	SharedMemoryBankConflictTest2<<<dimGrid2D, dimBlock2D>>>(matrix_Dev, matrixTestA_Dev, row, col);

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	std::cout << "\nTest2 ������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	hipEventRecord(start_GPU, 0);

	SharedMemoryBankConflictTest2_Pad<<<dimGrid2D, dimBlock2D>>>(matrix_Dev, matrixTestB_Dev, row, col);

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	std::cout << "Test2 ������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	hipEventRecord(start_GPU, 0);

	SharedMemoryBankConflictTest2_Across<<<dimGrid2D, dimBlock2D>>>(matrix_Dev, matrixTestC_Dev, row, col);

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	std::cout << "Test2 ������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	hipMemcpy(matrixTestA_Host, matrixTestA_Dev, row * col * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(matrixTestB_Host, matrixTestB_Dev, row * col * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(matrixTestC_Host, matrixTestC_Dev, row * col * sizeof(float), hipMemcpyDeviceToHost);

	std::cout << "����Ƿ���ͬ�� A-B : " << IsEqualMatrix(matrixTestA_Host, matrixTestB_Host, row, col)
	                     << "    A-C : " << IsEqualMatrix(matrixTestA_Host, matrixTestC_Host, row, col) << std::endl;

	hipEventDestroy(start_GPU);
	hipEventDestroy(end_GPU);

	hipFree(matrixTestA_Dev);
	hipFree(matrixTestB_Dev);
	hipFree(matrixTestC_Dev);
	free(matrixTestA_Host);
	free(matrixTestB_Host);
	free(matrixTestC_Host);
}