#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include "Test1.h"
#include "Test2.h"
#include "Test3.h"

#define BLOCKSIZE 32

using namespace std;

/**
* ���ܣ����� 0-1 ���������
* �����matrix �������������
* ���룺row ����
* ���룺col ����
*/
__global__ void GenerateMatrixCUDA(float *matrix, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	hiprandState s;
	hiprand_init(index, 0, 0, &s);

	if (x_id < col && y_id < row)
	{
		matrix[index] = hiprand_uniform(&s);
	}
	//if (x_id < 4 && y_id < 4)
	//	printf("x_id = %d, y_id = %d, value = %f.\n", x_id, y_id, matrix[index]);
}


int main()
{
	const int row = 1024;
	const int col = 1024;

	float *matrix_Dev;
	hipMalloc((void**)&matrix_Dev, row * col * sizeof(float));

	dim3 dimBlock2D(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid2D((col+ BLOCKSIZE - 1) / dimBlock2D.x, (row + BLOCKSIZE - 1) / dimBlock2D.y);
	GenerateMatrixCUDA<<<dimGrid2D, dimBlock2D>>>(matrix_Dev, row, col);

	/* ʵ�� 1 */
	Test1(matrix_Dev, row, col);

	/* ʵ�� 2 */
	Test2(matrix_Dev, row, col);

	/* ʵ�� 3 */
	Test3(matrix_Dev, row, col);

	hipFree(matrix_Dev);

	hipDeviceReset();
	return 0;
}