#include "hip/hip_runtime.h"
#include "Test1.h"
#include "IsEqualMatrix.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>

#define BLOCKSIZE 32

/**
* ���ܣ�ÿ�����ڵ�ÿ��Ԫ�ؼ���Ŀ���һ�еĺ�
* ˵��������ÿ�еļ�����ȫ��ͬ�����һ�ͬʱ����һ�� bank
* ���룺matrix �������
* �����matrixTest �������
* ���룺row ����
* ���룺col ����
*/
__global__ void SharedMemoryBankConflictTest1(float *matrix, float *matrixTest, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	__shared__ float sData[BLOCKSIZE][BLOCKSIZE];


	if (x_id < col && y_id < row)
	{
		sData[threadIdx.y][threadIdx.x] = matrix[index];
		__syncthreads();

		float data = 0.0f;
		for (int i = 0; i < 1; i++)
		{
			for (int j = 0; j < BLOCKSIZE; j++)
			{
				data += sData[threadIdx.x][j];
			}
		}
		matrixTest[index] = data;

	}
}


/**
* ���ܣ�ÿ�����ڵ�ÿ��Ԫ�ؼ���Ŀ���һ�еĺ�
* ˵��������ÿ�еļ�����ȫ��ͬ�����ڶ�����һ��������ÿһ�еĻ�ͬʱ���ʲ�ͬ�� bank
* ���룺matrix �������
* �����matrixTest �������
* ���룺row ����
* ���룺col ����
*/
__global__ void SharedMemoryBankConflictTest1_Pad(float *matrix, float *matrixTest, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	__shared__ float sData[BLOCKSIZE][BLOCKSIZE+1];


	if (x_id < col && y_id < row)
	{
		sData[threadIdx.y][threadIdx.x] = matrix[index];
		__syncthreads();

		float data = 0.0f;
		for (int i = 0; i < 1; i++)
		{
			for (int j = 0; j < BLOCKSIZE; j++)
			{
				data += sData[threadIdx.x][j];
			}
		}
		matrixTest[index] = data;

	}
}

/**
* ���ܣ���һ������ʾ�������� bank conflict
* ���룺matrix_Dev �������
* ���룺row ����
* ���룺col ����
*/
void Test1(float *matrix_Dev, const int row, const int col)
{
	float *matrixTestA_Dev, *matrixTestB_Dev;
	hipMalloc((void**)&matrixTestA_Dev, row * col * sizeof(float));
	hipMemset(matrixTestA_Dev, 0, row * col * sizeof(float));
	hipMalloc((void**)&matrixTestB_Dev, row * col * sizeof(float));
	hipMemset(matrixTestB_Dev, 0, row * col * sizeof(float));

	float *matrixTestA_Host = (float *)malloc(row * col * sizeof(float));
	float *matrixTestB_Host = (float *)malloc(row * col * sizeof(float));

	dim3 dimBlock2D(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid2D((col+ BLOCKSIZE - 1) / dimBlock2D.x, (row + BLOCKSIZE - 1) / dimBlock2D.y);

	// ��¼ʱ��
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	hipEventRecord(start_GPU, 0);

	SharedMemoryBankConflictTest1<<<dimGrid2D, dimBlock2D>>>(matrix_Dev, matrixTestA_Dev, row, col);

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	std::cout << "Test1 ������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	hipEventRecord(start_GPU, 0);

	SharedMemoryBankConflictTest1_Pad<<<dimGrid2D, dimBlock2D>>>(matrix_Dev, matrixTestB_Dev, row, col);

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	std::cout << "Test1 ������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	hipMemcpy(matrixTestA_Host, matrixTestA_Dev, row * col * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(matrixTestB_Host, matrixTestB_Dev, row * col * sizeof(float), hipMemcpyDeviceToHost);

	std::cout << "����Ƿ���ͬ��" << IsEqualMatrix(matrixTestA_Host, matrixTestB_Host, row, col) << std::endl;

	hipEventDestroy(start_GPU);
	hipEventDestroy(end_GPU);

	hipFree(matrixTestA_Dev);
	hipFree(matrixTestB_Dev);
	free(matrixTestA_Host);
	free(matrixTestB_Host);
}