#include "hip/hip_runtime.h"
#include "CountingSortCUDA.h"
#include "MyCuda.h"
#include <iostream>
#include <vector>

using namespace std;

/************************************************************************/
/*��������CountingSortKernelOne
/*��  �ܣ����������һ����C[i]ΪA��Ԫ��i�ĸ���
/*����ֵ����
/************************************************************************/
__global__ void CountingSortKernelOne(int *A, int *C, int iSize)
{
	// ��������
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������

	if (x_id < iSize)
	{
		int iIdx = A[x_id];
		atomicAdd(&C[iIdx], 1);
	}
}

/************************************************************************/
/*��������CountingSortKernelTwo
/*��  �ܣ���������ڶ����˺���������һ�ι�Լ
/*����ֵ����
/************************************************************************/
__global__ void CountingSortKernelTwo(int *C, int k, int iSize)
{
	// ��������
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������

	x_id++;

	if (x_id <= k)
	{
		int iRe = x_id & (iSize - 1);
		if (0 == iRe)
			iRe = iSize;

		if (iRe > (iSize >> 1))
		{
			x_id--;
			C[x_id] = C[x_id] + C[x_id + (iSize >> 1) - iRe];
		}
	}
}

/************************************************************************/
/*��������CountingSortTwo
/*��  �ܣ���������ڶ�������C��������ۻ��͹�Լ
/*����ֵ����
/************************************************************************/
void CountingSortTwo(dim3 dimGridK, dim3 dimBlockK, int *devC, int k)
{
	int iSize = 1;
	while (iSize <= k)
	{
		iSize = iSize << 1;
		CountingSortKernelTwo<<<dimGridK, dimBlockK>>>(devC, k, iSize);
	}
}

/************************************************************************/
/*��������CountingSortKernelThree
/*��  �ܣ��������������������C������±������������
/*����ֵ����
/************************************************************************/
__global__ void CountingSortKernelThree(int *B, int *C, int k)
{
	// ��������
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������

	if (x_id <= k)
	{
		int iStart = 0;
		if (0 != x_id)
			iStart = C[x_id - 1];

		for (int i = iStart; i < C[x_id]; i++)
		{
			B[i] = x_id;
		}
	}
}

/************************************************************************/
/*��������CountingSort
/*��  �ܣ�CUDA������������A�����B
/*����ֵ����
/************************************************************************/
void CountingSortCUDA(int *A, int *B, const int SIZE, const int k)
{
	vector<int> C(k + 1, 0); // ����C���鲢��ʼ��

	int *devA;
	int *devB;
	int *devC;

	hipMalloc((void**)&devA, sizeof(int) * SIZE);
	hipMalloc((void**)&devB, sizeof(int) * SIZE);
	hipMalloc((void**)&devC, sizeof(int) * (k + 1));

	hipMemcpy(devA, A, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemset(devB, 0, sizeof(int) * SIZE);
	hipMemset(devC, 0, sizeof(int) * (k + 1));

	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid((SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 dimGridK((k + BLOCK_SIZE - 1) / BLOCK_SIZE);

	// C�������ӦA��Ԫ�ص�ֵ��ͳ��A��Ԫ�صĸ���
	CountingSortKernelOne<<<dimGrid, dimBlock>>>(devA, devC, SIZE);

	// C�м����ۼӣ���ʱA[i]��ӦC�е�A[i]��
	// C[i]��������i��Ԫ�ظ���
	CountingSortTwo(dimGridK, dimBlock, devC, k + 1);


	// ��C[A[i]]���Ԫ�ؼ�ΪA[i]��B�е���ȷλ��
	// �����±�Ӧ��0��ʼ�������C[A[j]]��ȥ1
	// C[i]����С�ڵ���i��Ԫ�ظ���
	CountingSortKernelThree<<<dimGridK, dimBlock>>>(devB, devC, k);

	hipMemcpy(B, devB, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
}