#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <time.h>
#include <fstream>

#include "MyCuda.h"
#include "GenRandN.h"
#include "CountingSortCPU.h"
#include "IsOrder.h"
#include "CountingSortCUDA.h"

using namespace std;


/************************************************************************/
/*��������PrintfArray
/*��  �ܣ��������
/*����ֵ����
/************************************************************************/
void PrintfArray(int *A, const int SIZE)
{
	for (int i = 0; i < SIZE; ++i)
		cout << A[i] << " ";

	cout << endl;
}



const int N(80000000); // ���峣��
const int maxNum(10000000); // ���峣��

int main()
{
	vector<int> dataA(N, 0);
	vector<int> dataCPU(N, 0);
	vector<int> dataCUDA(N, 0);

	GenRandN(&dataA[0], N, maxNum); // �����������

	clock_t startTime, endTime;
	startTime = clock();

	CountingSort(&dataA[0], &dataCPU[0], N, maxNum); // ��������

	endTime = clock();
	cout<< "Run time: " << (double)(endTime - startTime) / CLOCKS_PER_SEC << "S" << endl;


	startTime = clock();

	CountingSortCUDA(&dataA[0], &dataCUDA[0], N, maxNum); // ��������

	endTime = clock();
	cout<< "Run time: " << (double)(endTime - startTime) / CLOCKS_PER_SEC << "S" << endl;


	// �ж������Ƿ�׼ȷ
	cout << IsOrder(&dataCPU[0], N) << endl;
	cout << IsOrder(&dataCUDA[0], N) << endl;
	cout << dataCPU[N-1] << " " << dataCUDA[N-1] << endl;

	return 0;
}

