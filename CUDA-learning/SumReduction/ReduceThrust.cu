#include "MyCuda.h"

#include <thrust\device_vector.h>
#include <thrust\host_vector.h>
#include <thrust\device_ptr.h>

/**
* ����reduce1������ʱ��
* fMatrix_Host  ����ͷָ��
* iRow          ��������
* iCol          ��������
* @return       ��
*/
float RuntimeOfReduceThrust(float *fMatrix_Host, const int iRow, const int iCol)
{
	// ������ά���Ƿ���ȷ
	if (iRow <= 0 || iCol <= 0)
	{
		std::cout << "The size of the matrix is error!" << std::endl;
		return 0.0;
	}

	float *fMatrix_Device; // ָ���豸�Դ�
	int iMatrixSize = iRow * iCol; // ����Ԫ�ظ���

	hipMalloc((void**)&fMatrix_Device, iMatrixSize * sizeof(float)); // ���Դ���Ϊ���󿪱ٿռ�
	hipMemcpy(fMatrix_Device, fMatrix_Host, iMatrixSize * sizeof(float), hipMemcpyHostToDevice); // �����ݿ������Դ�

	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;

	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	thrust::device_ptr<float> dev_ptr(fMatrix_Device);
	float thrustResult = thrust::reduce(dev_ptr, dev_ptr + size_t(iMatrixSize), (float)0, thrust::plus<float>());

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);
	hipEventDestroy(start_GPU);
	hipEventDestroy(end_GPU);

	std::cout << "GPU������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	hipFree(fMatrix_Device);

	return thrustResult;
}
