#include "hip/hip_runtime.h"
#include "MyCuda.h"

/**
* �Ż��������� reduce0 �� reduce1 �д��ڵ� warp ��֧���󣬽�һ�����Ч��
* �Ż������ reduce2 ���߳������ʵ͵����⣬�����ݿ����������ڴ�֮ǰ������
*       ִ��һ�κϲ�����һ������߳�������
* ���⣺1.��ÿ�����е�ʣ��Ԫ��С�ڵ���32��ʱ����Ȼ��ִ��ͬ������������ʵ����ÿ��warp����Ĭ
*       ��ͬ���ģ����ö����ͬ��������
* globalInputData  �������ݣ�λ��ȫ���ڴ�
* globalOutputData ������ݣ�λ��ȫ���ڴ�
*/
__global__ void reduce3(float *globalInputData, float *globalOutputData, unsigned int n)
{
	__shared__ float sdata[BLOCK_SIZE];

	// ��������
	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x*(blockDim.x * 2) + threadIdx.x;
	unsigned int indexWithOffset = index + blockDim.x;

	if (index >= n)
	{
		sdata[tid] = 0;
	}
	else if (indexWithOffset >= n)
	{
		sdata[tid] = globalInputData[index];
	}
	else
	{
		sdata[tid] = globalInputData[index] + globalInputData[indexWithOffset];
	}
	__syncthreads();

	// �ڹ����ڴ��ж�ÿһ������й�Լ����
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}

		__syncthreads();
	}

	// �Ѽ������ӹ����ڴ�д��ȫ���ڴ�
	if (tid == 0)
	{
		globalOutputData[blockIdx.x] = sdata[0];
	}
}

/**
* ����reduce3������ʱ��
* fMatrix_Host  ����ͷָ��
* iRow          ��������
* iCol          ��������
* @return       ��
*/
float RuntimeOfReduce3(float *fMatrix_Host, const int iRow, const int iCol)
{
	// ������ά���Ƿ���ȷ
	if (iRow <= 0 || iCol <= 0)
	{
		std::cout << "The size of the matrix is error!" << std::endl;
		return 0.0;
	}

	float *fReuslt = (float*)malloc(sizeof(float));;
	float *fMatrix_Device; // ָ���豸�Դ�
	int iMatrixSize = iRow * iCol; // ����Ԫ�ظ���

	HANDLE_ERROR(hipMalloc((void**)&fMatrix_Device, iMatrixSize * sizeof(float))); // ���Դ���Ϊ���󿪱ٿռ�
	HANDLE_ERROR(hipMemcpy(fMatrix_Device, fMatrix_Host, iMatrixSize * sizeof(float), hipMemcpyHostToDevice)); // �����ݿ������Դ�

	// ��¼��ʼʱ��
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;

	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	for (int i = 1, int iNum = iMatrixSize; i < iMatrixSize; i = 2 * i*BLOCK_SIZE)
	{
		int iBlockNum = (iNum + (2 * BLOCK_SIZE) - 1) / (2 * BLOCK_SIZE);
		reduce3<<<iBlockNum, BLOCK_SIZE>>>(fMatrix_Device, fMatrix_Device, iNum);
		iNum = iBlockNum;
	}

	HANDLE_ERROR(hipMemcpy(fReuslt, fMatrix_Device, sizeof(float), hipMemcpyDeviceToHost)); // �����ݿ������ڴ�

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);
	hipEventDestroy(start_GPU);
	hipEventDestroy(end_GPU);

	std::cout << "GPU������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	HANDLE_ERROR(hipFree(fMatrix_Device));// �ͷ��Դ�ռ�

	return fReuslt[0];
}