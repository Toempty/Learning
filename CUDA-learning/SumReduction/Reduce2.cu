#include "hip/hip_runtime.h"
#include "MyCuda.h"

/**
* ��GPU���� Sequential Addressing ���ϼ�������ĺ�
* CUDA�ж����ݵ�������ȡЧ��Ҫ��������ʽ�ߡ����������һ���Ż���Ҫ�ǽ�ȡַ��ʽ��Ϊ�����ġ�
* �Ż��������� reduce0 �� reduce1 �д��ڵ� warp ��֧���󣬽�һ�����Ч��
* ���⣺1.��ÿ�����е�ʣ��Ԫ��С�ڵ���32��ʱ����Ȼ��ִ��ͬ������������ʵ����ÿ��warp����Ĭ
*       ��ͬ���ģ����ö����ͬ��������
*       2.ִ��һ�κϲ�֮��ͻ���һ����̴߳�������״̬���߳�������ƫ��
* globalInputData  �������ݣ�λ��ȫ���ڴ�
* globalOutputData ������ݣ�λ��ȫ���ڴ�
*/
__global__ void reduce2(float *globalInputData, float *globalOutputData, unsigned int n)
{
	__shared__ float sdata[BLOCK_SIZE];

	// ��������
	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

	// ���ݶ��빲���ڴ�
	sdata[tid] = (index < n) ? globalInputData[index] : 0; // ������Χ����0

	__syncthreads(); // ͬ�����ȴ�ͬһ��block�ڵ����ݶ������������ڴ�

	// �ڹ����ڴ��ж�ÿһ������й�Լ����
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}

		__syncthreads();
	}

	// �Ѽ������ӹ����ڴ�д��ȫ���ڴ�
	if (tid == 0)
	{
		globalOutputData[blockIdx.x] = sdata[0];
	}
}

/**
* ����reduce2������ʱ��
* fMatrix_Host  ����ͷָ��
* iRow          ��������
* iCol          ��������
* @return       ��
*/
float RuntimeOfReduce2(float *fMatrix_Host, const int iRow, const int iCol)
{
	// ������ά���Ƿ���ȷ
	if (iRow <= 0 || iCol <= 0)
	{
		std::cout << "The size of the matrix is error!" << std::endl;
		return 0.0;
	}

	float *fReuslt = (float*)malloc(sizeof(float));;
	float *fMatrix_Device; // ָ���豸�Դ�
	int iMatrixSize = iRow * iCol; // ����Ԫ�ظ���

	HANDLE_ERROR(hipMalloc((void**)&fMatrix_Device, iMatrixSize * sizeof(float))); // ���Դ���Ϊ���󿪱ٿռ�
	HANDLE_ERROR(hipMemcpy(fMatrix_Device, fMatrix_Host, iMatrixSize * sizeof(float), hipMemcpyHostToDevice)); // �����ݿ������Դ�

	// ��¼��ʼʱ��
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;

	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	for (int i = 1, int iNum = iMatrixSize; i < iMatrixSize; i = i*BLOCK_SIZE)
	{
		int iBlockNum = (iNum + BLOCK_SIZE - 1) / BLOCK_SIZE;
		reduce2 << <iBlockNum, BLOCK_SIZE >> >(fMatrix_Device, fMatrix_Device, iNum);
		iNum = iBlockNum;
	}

	HANDLE_ERROR(hipMemcpy(fReuslt, fMatrix_Device, sizeof(float), hipMemcpyDeviceToHost)); // �����ݿ������ڴ�

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);
	hipEventDestroy(start_GPU);
	hipEventDestroy(end_GPU);

	std::cout << "Reduce2 ������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	HANDLE_ERROR(hipFree(fMatrix_Device));// �ͷ��Դ�ռ�

	return fReuslt[0];
}
