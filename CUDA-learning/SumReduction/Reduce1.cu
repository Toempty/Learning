#include "hip/hip_runtime.h"
#include "MyCuda.h"

/**
* ��GPU���� Interleaved Addressing ���ϼ�������ĺ�
* Interleaved Addressing �ĺ���˼�����ڽ���Ѱַ�������͵���״ģ��
* �Ż����Ľ� Reduce0 ��ȡ�����Ч�ʽϵ͵�����
* ���⣺ÿ��warp�ж�����ַ�֧��Ч�ʵ�
* globalInputData  �������ݣ�λ��ȫ���ڴ�
* globalOutputData ������ݣ�λ��ȫ���ڴ�
*/
__global__ void reduce1(float *globalInputData, float *globalOutputData, unsigned int n)
{
	__shared__ float sdata[BLOCK_SIZE];

	// ��������
	unsigned int tid = threadIdx.x;
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

	// ���ݶ��빲���ڴ�
	sdata[tid] = (index < n) ? globalInputData[index] : 0; // ������Χ����0

	__syncthreads(); // ͬ�����ȴ�ͬһ��block�ڵ����ݶ������������ڴ�

	// �ڹ����ڴ��ж�ÿһ������й�Լ����
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{
		index = 2 * s * tid;

		if (index < blockDim.x)
		{
			sdata[index] += sdata[index + s];
		}

		__syncthreads();
	}

	// �Ѽ������ӹ����ڴ�д��ȫ���ڴ�
	if (tid == 0)
	{
		globalOutputData[blockIdx.x] = sdata[0];
	}
}

/**
* ����reduce1������ʱ��
* fMatrix_Host  ����ͷָ��
* iRow          ��������
* iCol          ��������
* @return       ��
*/
float RuntimeOfReduce1(float *fMatrix_Host, const int iRow, const int iCol)
{
	// ������ά���Ƿ���ȷ
	if (iRow <= 0 || iCol <= 0)
	{
		std::cout << "The size of the matrix is error!" << std::endl;
		return 0.0;
	}

	float *fReuslt = (float*)malloc(sizeof(float));;
	float *fMatrix_Device; // ָ���豸�Դ�
	int iMatrixSize = iRow * iCol; // ����Ԫ�ظ���

	HANDLE_ERROR(hipMalloc((void**)&fMatrix_Device, iMatrixSize * sizeof(float))); // ���Դ���Ϊ���󿪱ٿռ�
	HANDLE_ERROR(hipMemcpy(fMatrix_Device, fMatrix_Host, iMatrixSize * sizeof(float), hipMemcpyHostToDevice)); // �����ݿ������Դ�

	// ��¼��ʼʱ��
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;

	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	for (int i = 1, int iNum = iMatrixSize; i < iMatrixSize; i = i*BLOCK_SIZE)
	{
		int iBlockNum = (iNum + BLOCK_SIZE - 1) / BLOCK_SIZE;
		reduce1<<<iBlockNum, BLOCK_SIZE>>>(fMatrix_Device, fMatrix_Device, iNum);
		iNum = iBlockNum;
	}

	HANDLE_ERROR(hipMemcpy(fReuslt, fMatrix_Device, sizeof(float), hipMemcpyDeviceToHost)); // �����ݿ������ڴ�

	// ��ʱ����
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);
	hipEventDestroy(start_GPU);
	hipEventDestroy(end_GPU);

	std::cout << "Reduce1 ������ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	HANDLE_ERROR(hipFree(fMatrix_Device));// �ͷ��Դ�ռ�

	return fReuslt[0];
}
