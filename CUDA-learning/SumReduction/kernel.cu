#include "MyCuda.h"
#include "CpuMatrix.h"

using namespace std;

int main(void)
{
	const int iRow = 11000;
	const int iCol = 11000;

	float *fMatrix_Host = (float*)malloc(iRow * iCol * sizeof(float)); // ���ٿռ�; // ָ�������ڴ�
	GenerateMatrix(fMatrix_Host, iRow, iCol); // �����������

	float thrustResult = RuntimeOfReduceThrust(fMatrix_Host, iRow, iCol);;
	cout << thrustResult << endl;

	float fReduce0Result = RuntimeOfReduce0(fMatrix_Host, iRow, iCol);
	cout << fReduce0Result << endl;

	float fReduce1Result = RuntimeOfReduce1(fMatrix_Host, iRow, iCol);
	cout << fReduce1Result << endl;

	float fReduce2Result = RuntimeOfReduce2(fMatrix_Host, iRow, iCol);
	cout << fReduce2Result << endl;

	float fReduce3Result = RuntimeOfReduce3(fMatrix_Host, iRow, iCol);
	cout << fReduce3Result << endl;

	float fReduce4Result = RuntimeOfReduce4(fMatrix_Host, iRow, iCol);
	cout << fReduce4Result << endl;

	float fCpuResult = RunTimeOfCpu(fMatrix_Host, iRow, iCol);
	cout << fCpuResult << endl;

	free(fMatrix_Host);

	system("pause");
	return 0;
}