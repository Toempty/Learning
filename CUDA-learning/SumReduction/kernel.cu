#include "MyCuda.h"
#include "CpuMatrix.h"

using namespace std;

int main(void)
{
	const int iRow = 1000;
	const int iCol = 1000;

	float *fMatrix_Host = (float*)malloc(iRow * iCol * sizeof(float)); // ���ٿռ�; // ָ�������ڴ�
	GenerateMatrix(fMatrix_Host, iRow, iCol); // �����������

	float thrustResult = RuntimeOfReduceThrust(fMatrix_Host, iRow, iCol);;
	cout << thrustResult << endl;

	float fReduce0Result = RuntimeOfReduce0(fMatrix_Host, iRow, iCol);
	cout << fReduce0Result << endl;

	float fReduce1Result = RuntimeOfReduce1(fMatrix_Host, iRow, iCol);
	cout << fReduce1Result << endl;

	float fReduce2Result = RuntimeOfReduce2(fMatrix_Host, iRow, iCol);
	cout << fReduce2Result << endl;

	float fReduce4Result = RuntimeOfReduce3(fMatrix_Host, iRow, iCol);
	cout << fReduce4Result << endl;

	float fReduce5Result = RuntimeOfReduce4(fMatrix_Host, iRow, iCol);
	cout << fReduce5Result << endl;

	float fCpuResult = RunTimeOfCpu(fMatrix_Host, iRow, iCol);
	cout << fCpuResult << endl;

	free(fMatrix_Host);

	system("pause");
	return 0;
}